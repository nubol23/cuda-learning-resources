
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/driver_types.h>
#include <hiprand/hiprand_kernel.h>

// KERNEL
__global__ void square(float *d_out, float *d_in){
    int idx = threadIdx.x;
    float f = d_in[idx];
//    d_out[idx] = f*f;
    d_out[idx] = f*f*f;
}

int main(){
//    const int ARRAY_SIZE = 64;
    const int ARRAY_SIZE = 96;
    const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

    //input array on host
    float *h_in = (float*) std::malloc(ARRAY_BYTES);
    for (int i = 0; i < ARRAY_SIZE; i++){
        h_in[i] = float(i);
    }

    float *h_out = (float*) std::malloc(ARRAY_BYTES);

    // GPU pointers
    float * d_in;
    float * d_out;

    // Allocate GPU
    hipMalloc((void **) &d_in, ARRAY_BYTES);
    hipMalloc((void **) &d_out, ARRAY_BYTES);

    // Move from cpu to gpu
    hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

    // Launch a kernel
    square<<<1, ARRAY_SIZE>>>(d_out, d_in);

    // copy back to cpu
    hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

    // Print array
    for (int i = 0; i < ARRAY_SIZE; i++){
        printf("%.0f", h_out[i]);
        printf(((i % 4) != 3) ? "\t": "\n");
    }

    hipFree(d_in);
    hipFree(d_out);

    std::free(h_in);
    std::free(h_out);

    return 0;
}